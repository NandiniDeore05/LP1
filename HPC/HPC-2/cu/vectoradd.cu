

#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
using namespace std;

__global__ void vectorAdd(int *a, int *b, int *result, int n) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid <= n) {
        result[tid] = a[tid] + b[tid];
    }
}

void print_array(int *a, int N) {
    for(int i=0; i<N; i++) {
        cout<<"  "<<a[i];
    }
    cout<<endl;
}

void init_array(int *a, int N) {
    for(int i=0; i<N; i++) {
        a[i] = rand()%10 + 1;
    }
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 8;           //24
    
    a = (int*)malloc(n * sizeof(n));
    b = (int*)malloc(n * sizeof(n));
    c = (int*)malloc(n * sizeof(n));

    int size = n * sizeof(int);
    
    hipMalloc(&a_dev, size);
    hipMalloc(&b_dev, size);
    hipMalloc(&c_dev, size);
    
    init_array(a, n);
    init_array(b, n);
    
    print_array(a, n);
    print_array(b, n);
        
    //cudaEvent_t start, end;
    //cudaEventCreate(&start);
    //cudaEventCreate(&end);
    
    hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);
    
    //int threads = 1024;
    //int blocks = (n+threads-1)/threads;
    
    //cudaEventRecord(start);
    
    //vectorAdd<<<blocks,threads>>>(a_dev, b_dev, c_dev, n);
    vectorAdd<<<1,1024>>>(a_dev, b_dev, c_dev, n);
    
    //cudaEventRecord(end);
    
    //cudaDeviceSynchronize();
       
    //float time = 0.0;
    //cudaEventElapsedTime(&time, start, end);
    
    hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost);
    
    cout<<"Results : "<<endl;
    print_array(c, n);
 
    //cout<<"Time elapsed : "<<time<<endl;
        
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
        
    return 0;
}
